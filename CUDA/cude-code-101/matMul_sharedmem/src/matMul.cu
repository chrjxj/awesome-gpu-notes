#include "hip/hip_runtime.h"
/********************************************************
CSci 360 Computer Architecture 3
Hunter College of the City University of New York Prof. Stewart Weiss
CUDA-based MATRIX MULTIPLICATION
This program demonstrates a common CUDA paradigm, as well as how CUDA relates to C/C++. It is not a main program, but a function that can
utilize the resources of the GPU to perform matrix multiplication.

BACKGROUND
CUDA extends the set of types of C/C++ with types whose names are of the form type1, type2, type3, and in some cases, type4
where "type" can be replaced by one of:
char, uchar, int, uint, long, ulong, short, ushort, or float
For example, char3, uint4, and ulong3 are all type names in CUDA.
These are called "vectors" in CUDA, but they are really C structs with members x, y, z, and w respectively. Thus, for example, if Point is declared as
int3 Point;
then Point.x, Point.y, and Point.z are the members of Point. These types all have constructors of the form make_type(arg1,...,arg4). Thus we can initialize a member as follows:
uint3 Dimensions_of_box(100,200,200);
The type dim3 is an extension of uint3. It specifies dimensions of things, but the constructor initializes all uninitialized components with the value 1. For example,
dim3 Dimensions_of_block(64, 64);
specifies a block that is 64 by 64 by 1 (because the z member is set to 1.)
ALGORITHM
This code is an implementation of matrix multiplication that takes advantage of the shared memory within each SM (streaming multiprocessor). It is more complex than the algorithm that uses just global device memory, for several inter-related reasons.
Each thread block is responsible for computing one square sub-matrix Csub of the product matrix C, and each thread within the block is responsible for computing one element of Csub.
Csub is equal to the product of two rectangular matrices:
the sub-matrix of A of dimension A_width x BLOCK_SIZE that has the same row indices as Csub, and
the sub-matrix of B of dimension BLOCK_SIZE x A_width that has the same column indices as Csub.
In order to fit into the device’s resources, these two rectangular matrices
are divided into as many square matrices of dimension BLOCK_SIZE as necessary and Csub is computed as the sum of the products of these square matrices.
Each of these products is computed by first loading the two corresponding
square matrices from global memory to shared memory, having each
thread compute one element of the product, and writing the resulting square sub-matrix back to global memory afterward.
To make the copying from global to shared memory efficient, each thread is responsible for copying a single element from each of the A and B matrices.
The copying is done in such a way to maximize the memory bandwidth, which will be explained below.
*********************************************************/
// CUDA libraries
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

// Include associated header file.
#include "../include/matMul.cuh"

// Thread block size
#define BLOCK_SIZE 16

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C);

double cpuSecond() {
    struct timeval tp;
    gettimeofday( & tp, NULL);
    return ((double) tp.tv_sec + (double) tp.tv_usec * 1.e-6);
}

#define BLOCK_SIZE 16 // Thread block size

/** Multiply(&A, %B, hA, wA, wB, &C) Host multiplication function, performs
 * C = A * B, where
 * hA is the height of A
 * wA is the width of A
 * wB is the width of B
 * and A, B, and C are linear arrays allocated by the calling program.
 * The arrays must have been allocated correctly; no chek is made. */
 
 void MatMul(const Matrix A, const Matrix B, Matrix C) {
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);    


    /* Compute the execution configuration assuming
    the matrix dimensions are multiples of BLOCK_SIZE   
    The dim3 declaration is used here. This specifies that dimBlock
    is BLOCK_SIZE x BLOCK_SIZE x 1 */
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    /* B.width/dimBlock.x is the length of a row of B divided by the horizontal size of a block, which yields the number of blocks in the horizontal dimension of the Grid.
    Similarly, A.height/dimBlock.y is the height of A divided by the vertical size of a block, which is the number of blocks vertically in the grid.
    These two values define the shape of the grid, i.e. the number of
    blocks horizontally and vertically. */
    uint gridwidth = B.width / dimBlock.x;
    uint gridheight = A.height / dimBlock.y;
    dim3 dimGrid(gridwidth, gridheight);

    /* Launch the kernel now. The syntax
    func<<< Dg, Db, Ns, S >>>(parameter_list)
    defines a kernel function with a parameter list that is executed on a specific configuration defined as follows:
    Dg is of type dim3 and specifies the dimension and size of
    the grid; Dg.x * Dg.y equals the number of blocks being launched; Dg.z must be equal to 1;
    Db is of type dim3 and specifies the dimension and size of
    each block, such that Db.x * Db.y * Db.z equals the number of threads per block;
    Ns is of type size_t and specifies the number of bytes in shared memory that is dynamically allocated per block for this call
    in addition to the statically allocated memory; Ns is an
    optional argument which defaults to 0;
    S is of type hipStream_t and specifies the associated stream;
    S is an optional argument which defaults to 0.
    The Muld() function will be run by every thread in every block of the grid. Thread blocks execute independently: they execute in any order, in parallel or in series, unpredictably. The function must be correct regardless of the order.
    */
    double iStart = cpuSecond();
    MatMulKernel<<< dimGrid, dimBlock>>> (d_A, d_B, d_C);
    hipDeviceSynchronize(); //wait for all GPU threads to complete
    double iElaps = cpuSecond() - iStart;
    printf("MatMulKernel execution time: %f\n", iElaps);
    
    // Copy result C from the device to host memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(d_A.elements); hipFree(d_B.elements); hipFree(d_C.elements);
}



/* Note 

the __global__ qualifier: this is a kernel function This also means that every thread executes this function.
When a thread executes this function, it has a specific thread id and block id. The thread id is the value of threadIdx, used below, and the block id is stored in blockIdx, used below.
threadIdx and blockIdx are of type unit3.


// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
    // Each thread computes one element of C
    // by accumulating results into Cvalue

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0 f;

    for (int i = 0; i < A.width; i++) {
        sum += A.elements[row * A.width + i] * B.elements[i * B.width + col];
    }

    C.elements[row * C.width + col] = sum;
}

*/

__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {    
    // Block index
    int block_col = blockIdx.x;
    int block_row = blockIdx.y;
    // Thread index
    int thread_col = threadIdx.x;
    int thread_row = threadIdx.y;
    /* Index of the first cell of the sub-matrix of A processed by the block
    of threads. There are A.width cells in a row and BLOCK_SIZE many rows in each sub-matrix. Therefore,
    A.width * BLOCK_SIZE * block_row
    is the total number of cells from the start of A to the leftmost cell of the first row of the horizontal set of sub-matrices of A processed by this block of threads.
    */
    int aBegin = A.width * BLOCK_SIZE * block_row;
    /* The index of the last cell in the row of A that starts at ABegin is aBegin plus A.width-1, since a row is A.width cells long.
     */
    int aEnd = aBegin + A.width - 1;
    /* Step size used to iterate through the sub-matrices of A
    The upper left corner of the next block of A is BLOCK_SIZE columns from the current block's corner, so the increment is just BLOCK_SIZE.
    */
    int aStep = BLOCK_SIZE;
    /* Index of the first sub-matrix of B processed by the block */
    int bBegin = BLOCK_SIZE * block_col;
    /* Step size used to iterate through the sub-matrices of B
    The upper left corner of the next block of B is BLOCK_SIZE rows below the upper left corner of the current block. Each row has B.width bytes, so the increment is BLOCK_SIZE * B.width. */
    int bStep = BLOCK_SIZE * B.width;
    /* The element of the block sub-matrix that is computed by the thread */
    float Csub = 0;
    /* Loop over all the sub-matrices of A and B required to compute the block sub-matrix */
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) {
        /* The __shared__ qualifier declares a variable that
        * resides in the shared memory space of a thread block, * has the lifetime of the block, and
        * is only accessible from the threads within the block.
        The As and Bs matrices declared below are in the shared memory of
        the block; As is for the sub-matrix of A, and Bs, a sub-matrix of B. */
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE]; /* Shared memory for the sub-matrix of B */

        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        /* The next step loads the matrices from global memory to shared memory; each thread loads one element of each matrix. There are two things
        to discuss about this: the logic and the performance.
        Performance Issue:
        The global memory access by all threads of a half-warp is coalesced into one or two memory transactions if it satisfies the following three conditions:
        1. Threads must access either
        4-byte words, resulting in one 64-byte memory transaction, or 8-byte words, resulting in one 128-byte memory transaction, or 16-byte words, resulting in two 128-byte memory transactions;
        2. All 16 words must lie in the same segment of size equal to the memory transaction size (or twice the memory transaction size when accessing 16-byte words);
        3. Threads must access the words in sequence, meaning that the kth thread in the half-warp must access the kth word.
        In the instructions below, each thread accesses a 4-byte word
        ( because it is a float) and 16 threads do this simultaneously,
        for a total of 64-bytes in each transaction. The words accessed by the threads are in sequence, so the memory accesses satisfy the conditions stated above, which implies that the accesses are coalesced and will require just two trips to global memory.
        Logic:
        The matrices As and Bs are shared by all threads in a given block. There are multiple blocks in a grid and hence multiple shared matrices at any given time. We focus on just one pair of these. Initially, the loop index variable, a, points to the upper-left
        corner of the rectangular sub-matrix of A that will contribute to the CSub result, and b points to the upper-left corner of the
        B sub-matrix.
        Remember that A and B are linear arrays in row-major order. The cell of A needed by the thread with id (row,col) is at an offset from a of A.width*row + col. The cell of B is similarly
        at an offset of B.width*row + col. The thread copies these two values into As[row,col] and Bs[rox,col] respectively.
        All threads in the thread block do this simultaneously, so the two arrays, As and Bs, are filled by the collection of threads in the
        thread block. These two arrays have the contents of just one BLOCK_SIZE by BLOCK_SIZE sub-matrix of each of A and B, not an entire row or column od A and B.
        In the next iteration of the loop, the a and b index variables
        point to the upper-left corners of two difference sub-matrices
        of A and B. The a index points to the sub-matrix to the immediate right of the preceding one, and b, to the one imediately below
        the preceding one.
        This continues until every sub-matrix in the row of A and in the
        column of B have been copied into As and Bs, one after the other. */
        As[thread_row][thread_col] = A.elements[a + A.width * thread_row + thread_col];
        Bs[thread_row][thread_col] = B.elements[b + B.width * thread_row + thread_col];

        /* The next step synchronizes to make sure the matrices are loaded: __syncthreads() is a barrier synchronization call; all threads in
        a single block wait here until every thread has made the call,
        at which point it returns in each thread. This guarantees that the BLOCK_SIZE by BLOCK_SIZE matrices As and Bs are completely filled.
        */
        __syncthreads();
        /* The next step is to compute the inner product of the row of As and column of Bs assigned to this thread, As[thread_row],
        and Bs[thread_col], i.e., the sum
        As[i][0]*Bs[0][j] + As[i][1]*Bs[1][j] + ... As[i][N-1]Bs[N-1][j]
        The loop below,
        for (int k = 0; k < BLOCK_SIZE; ++k)
        Csub += As[thread_row][k] * Bs[k][thread_col];
        adds to Csub the inner product of two vectors of size 16 corresponding to the row of a sub-matrix of A and a column of a sub-matrix of B. But each time the outer for-loop steps the index a to the next sub-matrix, this inner product is continued for the next sub-matrix to the right in A and the one below in B, so that when the loop completes, the inner product of the entire row of A and column of B has been computed and stored in Csub.
        For a device with compute capability 1.x, there are 16 banks in shared memory, each 4-bytes wide. A float is stored in one word of a bank.
        Remember that all threads within a single warp execute the same instruction at a time. Therefore, when a kernel has a loop such as
        for (int k = 0; k < BLOCK_SIZE; ++k)
        Csub += As[thread_row][k] * Bs[k][thread_col];
        each thread executes the instruction in the body simultaneously. Consider the assignment statement,
        Csub += As[thread_row][k] * Bs[k][thread_col];
        for fixed k. Each of the threads in a half-warp have the same
        value of thread_row, because a warp consists of 16 threads in
        row i followed by 16 threads in row i+1, for each i = 0,2,4,... 30.
        So the first half-warp accesses only row i, and the second half-warp accesses only row i+1. Put another way, all threads in a half-warp have the same value of thread_row. Thus, for a fixed value of k, As[thread_row][k] is the same memory location in each thread in a half-warp and will be read once and stored locally for each thread in a register.
        The values Bs[k][thread_col] are 16 successive 4-byte words,
        (because the arrays are stored in row-major order), stored
        in banks 0, 1, 2, ..., 15 of the shared memory. This is a fact of
        how shared memory is used by the GPU: successive 4-byte words are in successive banks. Therefore there is no bank conflict in the threads accessing these banks simultaneously. When simultaneous memory requests are to different banks, they are accessed concurrently. Therefore, the values Bs[k][thread_col] accessed by the 16 threads
        in a half-warp are accessed in just two clock cycles. */
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[thread_row][k] * Bs[k][thread_col];
        /* Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
         */
        __syncthreads();
    }
    /* Now that the loop has terminated, the value in Csub has to be copied back to global memory, to the corresponding cell of the C matrix. Each thread writes one element. The position is relative to the upper-left corner element of the C sub-matrix computed by this thread block. This sub-matrix is in position [block_row, block_col], so the upper-left corner is in the row given by
    B.width*BLOCK_SIZE*block_row at an offset from the 0 collumn of
    BLOCK_SIZE*block_col. */
    int c = B.width * BLOCK_SIZE * block_row + BLOCK_SIZE * block_col;
    /* The cell within this sub-matrix is in the row of the sub-matrix given by B.width*thread_row, and column thread_col. We assign Csub to this cell in the global array.
     */
     C.elements[c + B.width * thread_row + thread_col] = Csub;
}