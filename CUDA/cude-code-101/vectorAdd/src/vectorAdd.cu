#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// Include associated header file.
#include "../include/vectorAdd.cuh"

/**
 * Sample CUDA device function which adds an element from array A and array B.
 *
 */
__global__ void vectorAdd_kernel(double *A, double *B, double *C, int arraySize){
    // Get thread ID.
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread is within array bounds.
    if ( tid < arraySize ) {
        // Add a and b.
        C[tid] = A[tid] + B[tid];
    }
}



/**
 * Wrapper function for the CUDA kernel function.
 * @param A Array A.
 * @param B Array B.
 * @param C Sum of array elements A and B directly across.
 * @param arraySize Size of arrays A, B, and C.
 */
void vectorAdd(double *A, double *B, double *C, int arraySize) {

    // Initialize device pointers.
    double *d_A, *d_B, *d_C;

    // Allocate device memory.
    hipMalloc((void**) &d_A, arraySize * sizeof(double));
    hipMalloc((void**) &d_B, arraySize * sizeof(double));
    hipMalloc((void**) &d_C, arraySize * sizeof(double));

    // Transfer arrays a and b to device.
    hipMemcpy(d_A, A, arraySize * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, arraySize * sizeof(double), hipMemcpyHostToDevice);

    // Calculate blocksize and gridsize.
    dim3 blockSize(512, 1, 1);
    dim3 gridSize(512 / arraySize + 1, 1);

    // Launch CUDA kernel.
    vectorAdd_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, arraySize);

    // Copy result array c back to host memory.
    hipMemcpy(C, d_C, arraySize * sizeof(double), hipMemcpyDeviceToHost);
}











