// Include C++ header files.

#include <iostream>
#include <pthread.h>
#include <stdio.h>
#include <math.h> 

// Include local CUDA header files.
#include <hip/hip_runtime.h>



const int N = 1 << 20;

__global__ void kernel(float *x, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = sqrt(pow(3.14159,i));
    }
}

void *launch_kernel(void *dummy)
{
    float *data;
    hipMalloc(&data, N * sizeof(float));

    kernel<<<1, 64>>>(data, N);

    hipStreamSynchronize(0);

    return NULL;
}

int main()
{
    const int num_threads = 8;

    pthread_t threads[num_threads];

    for (int i = 0; i < num_threads; i++) {
        if (pthread_create(&threads[i], NULL, launch_kernel, 0)) {
            fprintf(stderr, "Error creating threadn");
            return 1;
        }
    }

    for (int i = 0; i < num_threads; i++) {
        if(pthread_join(threads[i], NULL)) {
            fprintf(stderr, "Error joining threadn");
            return 2;
        }
    }

    hipDeviceReset();

    return 0;
}
