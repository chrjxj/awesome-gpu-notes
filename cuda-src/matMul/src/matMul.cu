#include "hip/hip_runtime.h"
// CUDA libraries.
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <sys/time.h>

// Include associated header file.
#include "../include/matMul.cuh"

// Thread block size
#define BLOCK_SIZE 16


double cpuSecond() 
{
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec*1.e-6);
}


// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row*A.width + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row*A.width + col] = value;
}


// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col)
{
  Matrix Asub;
  Asub.width = BLOCK_SIZE;
  Asub.height = BLOCK_SIZE;
  Asub.elements = &A.elements[BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
    // Each thread computes one element of C
    // by accumulating results into Cvalue
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    float sum = 0.0f;

    for (int i = 0; i < A.width; i++) {
        sum += A.elements[row * A.width + i] * B.elements[i * B.width + col];
    }

    C.elements[row * C.width + col] = sum;
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
    // Load A and B to device memory
    Matrix d_A;
    d_A.width = A.width; d_A.height = A.height;
    size_t size = A.width * A.height * sizeof(float);
    hipMalloc(&d_A.elements, size);
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

    Matrix d_B;
    d_B.width = B.width; d_B.height = B.height;
    size = B.width * B.height * sizeof(float);
    hipMalloc(&d_B.elements, size);
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

    // Allocate C in device memory
    Matrix d_C;
    d_C.width = C.width; d_C.height = C.height;
    size = C.width * C.height * sizeof(float);
    hipMalloc(&d_C.elements, size);

    // Invoke kernel
    // *** bug in dim assignment ***
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
    

    double iStart = cpuSecond();
    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
    hipDeviceSynchronize();  //wait for all GPU threads to complete
    double iElaps = cpuSecond() - iStart;
    printf ("MatMulKernel execution time: %f\n", iElaps);
    // Read C from device memory
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A.elements);
    hipFree(d_B.elements);
    hipFree(d_C.elements);
}
